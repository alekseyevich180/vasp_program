// File: cuda_main.cu
// C/Fortran interface to CUDA C API.

// includes standard headers
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
// includes cuda headers
#include <hip/hip_runtime.h>
// includes project headers
#include "cuda_globals.h"

#ifdef __PARA
//#undef SEEK_SET  // remove compilation errors
//#undef SEEK_CUR  // with C++ binding of MPI
//#undef SEEK_END
#include <mpi.h>
#endif



// global variables
int NUM_STREAMS=0;              // number of CUDA streams
hipStream_t *stream;		// CUDA stream
double *d_reduce, *d_reduce1;	// arrays for parallel reduction
hipDoubleComplex *d_zreduce, *d_zreduce1;  // arrays for parallel reduction
devptr_t *d_ptrs, *d_ptrs1;	// arrays of device pointers
int nPE_, myPE_;


/******************************************************/
// CUDA C wrappers for init, used in VAMP

extern "C"
void cuda_init_(int *nstreams, int *nsim)
{
    int i;

    /* Get MPI Information */
#ifdef __PARA
    MPI_Comm_size(MPI_COMM_WORLD, &nPE_);
    MPI_Comm_rank(MPI_COMM_WORLD, &myPE_);
#else
    nPE_ = 1; myPE_ = 0;
#endif


    // check number of CUDA streams requested
    if(*nstreams<=0)
    {
        printf("Nstreams is %d\n", *nstreams);
        ERROR( "GPU Library", "Invalid number of CUDA streams:pick a number greater than zero!");
    }
    NUM_STREAMS=*nstreams;
    printf("creating %d CUDA streams...\n",NUM_STREAMS);

    // create CUDA streams
    stream=(hipStream_t*)malloc(NUM_STREAMS*sizeof(hipStream_t));
    for(i=0;i<NUM_STREAMS;i++)
	CUDA_ERROR( hipStreamCreate(&stream[i]), "Failed to create CUDA stream!" );

    // allocate parallel reduction arrays
    CUDA_ERROR( hipMalloc((void **)&d_zreduce,MAX_THREADS*sizeof(hipDoubleComplex)),
		"Failed to allocate device memory!" );
    CUDA_ERROR( hipMalloc((void **)&d_zreduce1,MAX_THREADS*sizeof(hipDoubleComplex)),
                "Failed to allocate device memory!" );
    // set parallel reduction arrays
    d_reduce = (double *)d_zreduce;
    d_reduce1 = (double *)d_zreduce1;

    // allocate device pointer arryas
    CUDA_ERROR( hipMalloc((void **)&d_ptrs,(*nsim)*sizeof(devptr_t)),
		"Failed to allocate device memory!" );
    CUDA_ERROR( hipMalloc((void **)&d_ptrs1,(*nsim)*sizeof(devptr_t)),
                "Failed to allocate device memory!" );
}

extern "C"
void cuda_destroy_(void)
{
    int i;
    // destroy CUDA streams
    for(i=0;i<NUM_STREAMS;i++)
        CUDA_ERROR( hipStreamDestroy(stream[i]), "Failed to destroy CUDA stream!" );
    free(stream);

    // free parallel reduction arrays
    CUDA_ERROR( hipFree(d_reduce), "Failed to allocate device memory!" );
    CUDA_ERROR( hipFree(d_reduce1), "Failed to allocate device memory!" );

    // free device pointer arrays
    CUDA_ERROR( hipFree(d_ptrs), "Failed to allocate device memory!" );
    CUDA_ERROR( hipFree(d_ptrs1), "Failed to allocate device memory!" );
}

// TODO: replace with init from exact exchange?
extern "C"
void cuda_mpi_init_(int *CudaDevice)
{
#ifndef EMULATION
    int deviceCount, gpu_rank;
    hipDeviceProp_t deviceProp;

    /* Get MPI Information */
#ifdef __PARA
    MPI_Comm_size(MPI_COMM_WORLD, &nPE_);
    MPI_Comm_rank(MPI_COMM_WORLD, &myPE_);
#else
    nPE_ = 1; myPE_ = 0;
#endif



    CUDA_ERROR( hipGetDeviceCount(&deviceCount), "No CUDA-supporting devices found!" );

    gpu_rank = (*CudaDevice)*deviceCount/nPE_;
    CUDA_ERROR( hipGetDeviceProperties(&deviceProp, gpu_rank),
		"Device does not support CUDA!" );
    if(deviceProp.major < 1)
    {
        printf( "CUDA ERROR: Devices does not support CUDA!\n");
        hipDeviceReset();
	exit(1);
    }
    printf("Using device %d (rank %d) : %s\n", gpu_rank,*CudaDevice,deviceProp.name);
    CUDA_ERROR(hipSetDevice(gpu_rank), "Failed to set the device!" );
#endif
}

/******************************************************/
// CUDA C wrappers for thread sync, in VASP

extern "C"
void cuda_device_reset_(void)
{
    printf("Reseting the CUDA device...\n");
    CUDA_ERROR( hipDeviceReset(), "Failed to reset the device!" );
}

// synchronze the device
extern "C"
void cuda_devicesynchronize_(char *msg)
{
    CUDA_ERROR( hipDeviceSynchronize(), msg );
}

// in fortran source
extern "C"
void threadsynchronize_(void)
{
    CUDA_ERROR( hipDeviceSynchronize(), "Failed to synchronize the device!" );
}

extern "C"
void cuda_streamsynchronize_(int *sid)
{
    hipStream_t st = CUDA_STREAM(*sid);  // CUDA stream
    CUDA_ERROR( hipStreamSynchronize(st), "Failed to synchronize the CUDA stream!" );
}

extern "C"
void cuda_all_stream_synchronize_(void)
{
    CUDA_ERROR( hipStreamSynchronize(0), "Failed to synchronize all CUDA streams!" );
}

/******************************************************/
